
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void invertKernel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < width * height * channels) {
            output[idx] = 255 - input[idx];
        }
    }
}
